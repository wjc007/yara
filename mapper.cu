// ==========================================================================
//                 SeqAn - The Library for Sequence Analysis
// ==========================================================================
// Copyright (c) 2013 NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of NVIDIA Corporation nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
// OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.
//
// ==========================================================================
// Author: Enrico Siragusa <enrico.siragusa@fu-berlin.de>
// ==========================================================================

// ============================================================================
// Prerequisites
// ============================================================================

// ----------------------------------------------------------------------------
// SeqAn headers
// ----------------------------------------------------------------------------

#include <seqan/basic.h>
#include <seqan/sequence.h>
#include <seqan/index.h>
#include <seqan/store.h>
#include <seqan/misc/misc_cuda.h>

// ----------------------------------------------------------------------------
// I/O and options
// ----------------------------------------------------------------------------

#include "tags.h"
#include "reads.h"
#include "genome.h"

// ----------------------------------------------------------------------------
// App headers
// ----------------------------------------------------------------------------

#include "types.h"
#include "misc.h"
//#include "options.h"
#include "mapper.h"
#include "mapper.cuh"

using namespace seqan;

// ============================================================================
// Functions
// ============================================================================

// ----------------------------------------------------------------------------
// Function assign()                                                  [FMIndex]
// ----------------------------------------------------------------------------
// NOTE(esiragusa): We do not assign the text to the device index!

namespace seqan {
template <typename TValue, typename TAlloc, typename TSSetSpec, typename TOccSpec, typename TSpec,
          typename TText2, typename TOccSpec2, typename TSpec2>
inline void
assign(Index<StringSet<thrust::device_vector<TValue, TAlloc>, TSSetSpec>, FMIndex<TOccSpec, TSpec> > & index,
       Index<TText2, FMIndex<TOccSpec2, TSpec2> > & source)
{
    cudaPrintFreeMemory();

    assign(indexSA(index), indexSA(source));
    assign(indexLF(index), indexLF(source));

    cudaPrintFreeMemory();
}
}

// --------------------------------------------------------------------------
// Function mapReads()
// --------------------------------------------------------------------------

void mapReads(Mapper<ExecDevice> & mapper, Options const & options)
{
    typedef typename Device<TGenomeIndex>::Type                 TDeviceIndex;
    typedef typename Device<TReadSeqs>::Type                    TDeviceReadSeqs;

    // Copy index to device.
    TDeviceIndex deviceIndex;
    assign(deviceIndex, mapper.index);

    // Copy read seqs to device.
    TDeviceReadSeqs deviceReadSeqs;
    assign(deviceReadSeqs, getSeqs(mapper.reads));

    // Wait for the copy to finish.
    hipDeviceSynchronize();

    // Map reads.
    _mapReads(mapper, options, deviceIndex, deviceReadSeqs);
}
